#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <cfilters.h>

#define CONV(l, c, nb_c) \
    (l) * (nb_c) + (c)

__global__ void apply_gray_filter_image_gpu_kernel(pixel *p, int width, int height)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j < width * height)
    {
        int moy;

        moy = (p[j].r + p[j].g + p[j].b) / 3;
        if (moy < 0)
            moy = 0;
        if (moy > 255)
            moy = 255;

        p[j].r = moy;
        p[j].g = moy;
        p[j].b = moy;
    }
}

__global__ void apply_blur_filter_image_gpu_kernel(pixel *p, pixel *p_new, int *end,
                                            int width, int height, int size, int threshold)
{
    int position = threadIdx.x + blockIdx.x * blockDim.x;
    int j = position / width;
    int k = position % width;

    if (k == 0 && j == 0)
    {
        *end = 1;
    }

    if (k >= size && k < width - size)
    {

        if (j >= size && j < height / 10 - size || j >= height * 0.9 + size && j < height - size)
        {
            int stencil_j, stencil_k;
            int t_r = 0;
            int t_g = 0;
            int t_b = 0;

            for (stencil_j = -size; stencil_j <= size; stencil_j++)
            {
                for (stencil_k = -size; stencil_k <= size; stencil_k++)
                {
                    t_r += p[CONV(j + stencil_j, k + stencil_k, width)].r;
                    t_g += p[CONV(j + stencil_j, k + stencil_k, width)].g;
                    t_b += p[CONV(j + stencil_j, k + stencil_k, width)].b;
                }
            }

            p_new[CONV(j, k, width)].r = t_r / ((2 * size + 1) * (2 * size + 1));
            p_new[CONV(j, k, width)].g = t_g / ((2 * size + 1) * (2 * size + 1));
            p_new[CONV(j, k, width)].b = t_b / ((2 * size + 1) * (2 * size + 1));
        }

        if (j >= height / 10 - size && j < height * 0.9 + size)
        {
            p_new[CONV(j, k, width)].r = p[CONV(j, k, width)].r;
            p_new[CONV(j, k, width)].g = p[CONV(j, k, width)].g;
            p_new[CONV(j, k, width)].b = p[CONV(j, k, width)].b;
        }
    }

    __threadfence();

    if (j >= 1 && j < height - 1 && k >= 1 && k < width - 1)
    {
        float diff_r;
        float diff_g;
        float diff_b;

        diff_r = (p_new[CONV(j, k, width)].r - p[CONV(j, k, width)].r);
        diff_g = (p_new[CONV(j, k, width)].g - p[CONV(j, k, width)].g);
        diff_b = (p_new[CONV(j, k, width)].b - p[CONV(j, k, width)].b);

        if (diff_r > threshold || -diff_r > threshold ||
            diff_g > threshold || -diff_g > threshold ||
            diff_b > threshold || -diff_b > threshold)
        {
            *end = 0;
        }
    }

    __threadfence();

    if (j >= 1 && j < height - 1 && k >= 1 && k < width - 1)
    {
        p[CONV(j, k, width)].r = p_new[CONV(j, k, width)].r;
        p[CONV(j, k, width)].g = p_new[CONV(j, k, width)].g;
        p[CONV(j, k, width)].b = p_new[CONV(j, k, width)].b;
    }
}

__global__ void apply_sobel_filter_image_gpu_kernel(pixel *p, pixel *p_new, int width, int height)
{
    int position = threadIdx.x + blockIdx.x * blockDim.x;
    int j = position / width;
    int k = position % width;

    if (j >= 1 && j < height - 1 && k >= 1 && k < width - 1)
    {
        int pixel_blue_no, pixel_blue_n, pixel_blue_ne;
        int pixel_blue_so, pixel_blue_s, pixel_blue_se;
        // int pixel_blue_o , pixel_blue  , pixel_blue_e ;
        int pixel_blue_o, pixel_blue_e;

        float deltaX_blue;
        float deltaY_blue;
        float val_blue;

        pixel_blue_no = p[CONV(j - 1, k - 1, width)].b;
        pixel_blue_n = p[CONV(j - 1, k, width)].b;
        pixel_blue_ne = p[CONV(j - 1, k + 1, width)].b;
        pixel_blue_so = p[CONV(j + 1, k - 1, width)].b;
        pixel_blue_s = p[CONV(j + 1, k, width)].b;
        pixel_blue_se = p[CONV(j + 1, k + 1, width)].b;
        pixel_blue_o = p[CONV(j, k - 1, width)].b;
        // pixel_blue = p[CONV(j, k - 1, width)].b;
        pixel_blue_e = p[CONV(j, k + 1, width)].b;

        deltaX_blue = -pixel_blue_no + pixel_blue_ne - 2 * pixel_blue_o + 2 * pixel_blue_e - pixel_blue_so + pixel_blue_se;

        deltaY_blue = pixel_blue_se + 2 * pixel_blue_s + pixel_blue_so - pixel_blue_ne - 2 * pixel_blue_n - pixel_blue_no;

        val_blue = sqrt(deltaX_blue * deltaX_blue + deltaY_blue * deltaY_blue) / 4;

        if (val_blue > 50)
        {
            p_new[CONV(j, k, width)].r = 255;
            p_new[CONV(j, k, width)].g = 255;
            p_new[CONV(j, k, width)].b = 255;
        }
        else
        {
            p_new[CONV(j, k, width)].r = 0;
            p_new[CONV(j, k, width)].g = 0;
            p_new[CONV(j, k, width)].b = 0;
        }
    }

    else
    {
        if (j < height && k < width)
        {
            p_new[CONV(j, k, width)] = p[CONV(j, k, width)];
        }
    }
}

extern "C"
{
    void apply_gray_filter_gif_gpu(animated_gif *image)
    {
        int i;
        int width = image->width[0];
        int height = image->height[0];
        int first_image_size = width * height;

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        pixel *output_image;
        hipMalloc(&output_image, first_image_size * sizeof(pixel));
        dim3 dimGrid(first_image_size / deviceProp.maxThreadsPerBlock + 1);
        dim3 dimBlock(deviceProp.maxThreadsPerBlock);

        for (i = 0; i < image->n_images; i++)
        {
            hipMemcpy(output_image, image->p[i], first_image_size * sizeof(pixel), hipMemcpyHostToDevice);
            apply_gray_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(output_image, width, height);
            hipMemcpy(image->p[i], output_image, first_image_size * sizeof(pixel), hipMemcpyDeviceToHost);
        }

        hipFree(output_image);
    }

    void apply_blur_filter_gif_gpu(animated_gif *image, int blur_size, int threshold)
    {
        int i;
        int width = image->width[0];
        int height = image->height[0];
        int end = 0;
        int first_image_size = width * height;

        int *end_device;

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        pixel *temp_image, *output_image;
        hipMalloc(&temp_image, first_image_size * sizeof(pixel));
        hipMalloc(&output_image, first_image_size * sizeof(pixel));
        hipMalloc(&end_device, sizeof(int));

        dim3 dimGrid(first_image_size / deviceProp.maxThreadsPerBlock + 1);
        dim3 dimBlock(deviceProp.maxThreadsPerBlock);

        for (i = 0; i < image->n_images; i++)
        {
            hipMemcpy(temp_image, image->p[i], first_image_size * sizeof(pixel), hipMemcpyHostToDevice);
            int n_iter = 0;
            end = 1;
            do
            {
                n_iter++;
                apply_blur_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(temp_image, output_image, end_device, width, height, blur_size, threshold);
                hipMemcpy(&end, end_device, sizeof(int), hipMemcpyDeviceToHost);
            } while (threshold > 0 && !end);
            hipMemcpy(image->p[i], output_image, first_image_size * sizeof(pixel), hipMemcpyDeviceToHost);
        }

        hipFree(temp_image);
        hipFree(output_image);
        hipFree(end_device);
    }

    void apply_sobel_filter_gif_gpu(animated_gif *image)
    {
        int i;
        int width = image->width[0];
        int height = image->height[0];
        int first_image_size = width * height;

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        pixel *temp_image, *output_image;
        hipMalloc(&temp_image, first_image_size * sizeof(pixel));
        hipMalloc(&output_image, first_image_size * sizeof(pixel));

        dim3 dimGrid(first_image_size / deviceProp.maxThreadsPerBlock + 1);
        dim3 dimBlock(deviceProp.maxThreadsPerBlock);

        for (i = 0; i < image->n_images; i++)
        {
            hipMemcpy(temp_image, image->p[i], first_image_size * sizeof(pixel), hipMemcpyHostToDevice);
            apply_sobel_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(temp_image, output_image, width, height);
            hipMemcpy(image->p[i], output_image, first_image_size * sizeof(pixel), hipMemcpyDeviceToHost);
        }

        hipFree(temp_image);
        hipFree(output_image);
    }
}

void apply_all_filters_gif_gpu(animated_gif *image, int blur_size, int threshold)
{
    int i;
    int width = image->width[0];
    int height = image->height[0];
    int end = 0;
    int first_image_size = width * height;

    int *end_device;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    dim3 dimGrid(10 * first_image_size / deviceProp.maxThreadsPerBlock + 1);
    dim3 dimBlock(deviceProp.maxThreadsPerBlock / 10);

    pixel *temp_image, *output_image;
    hipMalloc(&temp_image, first_image_size * sizeof(pixel));
    hipMalloc(&output_image, first_image_size * sizeof(pixel));
    hipMalloc(&end_device, sizeof(int));

    for (i = 0; i < image->n_images; i++)
    {
        hipMemcpy(temp_image, image->p[i], first_image_size * sizeof(pixel), hipMemcpyHostToDevice);
        apply_gray_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(temp_image, width, height);
        int n_iter = 0;
        end = 1;
        do
        {
            n_iter++;
            apply_blur_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(temp_image, output_image, end_device, width, height, blur_size, threshold);
            hipMemcpy(&end, end_device, sizeof(int), hipMemcpyDeviceToHost);
        } while (threshold > 0 && !end);
        apply_sobel_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(temp_image, output_image, width, height);
        hipMemcpy(image->p[i], output_image, first_image_size * sizeof(pixel), hipMemcpyDeviceToHost);
    }

    hipFree(temp_image);
    hipFree(output_image);
    hipFree(end_device);
}

void apply_all_filters_image_gpu(animated_gif *image, int rank, int blur_size, int threshold)
{
    int width = image->width[rank];
    int height = image->height[rank];
    int end = 0;
    int first_image_size = width * height;

    int *end_device;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    dim3 dimGrid(10 * first_image_size / deviceProp.maxThreadsPerBlock + 1);
    dim3 dimBlock(deviceProp.maxThreadsPerBlock / 10);

    pixel *temp_image, *output_image;
    hipMalloc(&temp_image, first_image_size * sizeof(pixel));
    hipMalloc(&output_image, first_image_size * sizeof(pixel));
    hipMalloc(&end_device, sizeof(int));

    hipMemcpy(temp_image, image->p[rank], first_image_size * sizeof(pixel), hipMemcpyHostToDevice);
    apply_gray_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(temp_image, width, height);
    int n_iter = 0;
    end = 1;
    do
    {
        n_iter++;
        apply_blur_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(temp_image, output_image, end_device, width, height, blur_size, threshold);
        hipMemcpy(&end, end_device, sizeof(int), hipMemcpyDeviceToHost);
    } while (threshold > 0 && !end);
    apply_sobel_filter_image_gpu_kernel<<<dimGrid, dimBlock>>>(temp_image, output_image, width, height);
    hipMemcpy(image->p[rank], output_image, first_image_size * sizeof(pixel), hipMemcpyDeviceToHost);

    hipFree(temp_image);
    hipFree(output_image);
    hipFree(end_device);
}